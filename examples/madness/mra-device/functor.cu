#include "hip/hip_runtime.h"
#include <iostream>
#include <cuda/std/atomic>
#include <hip/hip_runtime.h>


template <typename T>
void random_ints(T *arr, int size) {
    for (int i = 0; i < size; ++i) {
        arr[i] = rand() % 100; // Generate random int between 0 and 99
    }
}

template <typename T>
__device__ float cexp(T a) {
    return std::exp(a);
}

class functor {
    public:
        template <typename T>
        __device__ T operator()(T x) const{
            return cexp(x);
        }
};

template <typename T, class F>
__global__ void kernel(const F &f, T *x, T *y, int N){
    for (int i = 0; i < N; ++i) {
        y[i] = f(x[i]);
    }
}

int main(){
    int N = 10;
    float *a, *c, *da, *dc;
    functor f, *d_f;

    a = (float*)malloc(sizeof(float)*N); random_ints(a, N);
    c = (float*)malloc(sizeof(float)*N);

    hipMalloc(&d_f, sizeof(functor));
    hipMalloc(&da, sizeof(float)*N);
    hipMalloc(&dc, sizeof(float)*N);
    hipMemcpy(d_f, &f, sizeof(functor), hipMemcpyHostToDevice);
    hipMemcpy(da, a, sizeof(float)*N, hipMemcpyHostToDevice);
    hipMemcpy(dc, c, sizeof(float)*N, hipMemcpyHostToDevice);

    kernel<<<1,1>>>(*d_f, da, dc, N);

    hipMemcpy(c, dc, sizeof(float)*N, hipMemcpyDeviceToHost);

    for (int i = 0; i < N; ++i) {
        std::cout << c[i] << std::endl;
    }

    free(a); free(c);
    hipFree(da); hipFree(dc); hipFree(d_f);

    return 0;

}